
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple nvptx64-unknown-unknown | FileCheck %s
// RUN: %clang_cc1 -emit-llvm %s -o - -fcuda-is-device -triple amdgcn-amd-amdhsa | FileCheck %s --check-prefix=NOASCAST


// Make sure we emit the proper addrspacecast for llvm.used iff necessary.
// PR22383 exposed an issue where we were generating a bitcast instead of an
// addrspacecast.

// CHECK: @llvm.compiler.used = appending global [1 x ptr] [ptr addrspacecast (ptr addrspace(1) @a to ptr)], section "llvm.metadata"
// NOASCAST: @llvm.compiler.used = appending addrspace(1) global [1 x ptr addrspace(1)] [ptr addrspace(1) @a], section "llvm.metadata"
__attribute__((device)) __attribute__((__used__)) int a[] = {};
